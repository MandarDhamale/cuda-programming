#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N){

    //intialize global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < N){
        C[idx] = A[idx] + B[idx];
    }

}

int main(){


    const int N = 1024;
    size_t size = N * sizeof(float);

    //cpu variables

    float *h_A = (float*) malloc(size);
    float *h_B = (float*) malloc(size);
    float *h_C = (float*) malloc(size);

    //intializing the array 

    for(int i=0; i < N; i++){
        h_A[i] = (float) i;
        h_B[i] = (float) i;
    }

    //gpu variables

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // copy the arrays from host to device

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //defining thread & grid size 

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1)/threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // wait until the gpu is done 

    hipDeviceSynchronize();
    hipGetLastError();

    // copy the result back from gpu to cpu

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    //priting the reults to verify if it is correct

    for(int i=0; i<N; i++){
        printf("%f + %f = %f", h_A[i], h_B[i], h_C[i]);
        printf("\n");
    }

    //free up memory to avoid leaks 

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);


}